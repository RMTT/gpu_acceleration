#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <ctime>
#include "sobel.cuh"
#include "matrix_power.cuh"

using namespace cv;


int main() {
    Mat gray_img = imread("C:\\Users\\RMT\\Downloads\\6k.jpg", 0);
    namedWindow("origin", 0);
    imshow("origin", gray_img);
    int height = gray_img.rows;
    int width = gray_img.cols;

    Mat gauss_image;
    GaussianBlur(gray_img, gauss_image, Size(3, 3), 0, 0, BORDER_DEFAULT);

    Mat result_cpu(height, width, CV_8UC1, Scalar(0)), result_gpu(height, width, CV_8UC1, Scalar(0));

    clock_t start, end;
    start = clock();
    sobel(gauss_image, result_cpu, height, width);
    end = clock();

    namedWindow("result_cpu", 0);
    imshow("result_cpu", result_cpu);
    printf("cpu: %ldms\n", end - start);

    callCuda(gauss_image, result_gpu, height, width);

    waitKey(0);
    return 0;
}